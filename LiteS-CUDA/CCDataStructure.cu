#include "hip/hip_runtime.h"
#include "CCDataStructure.h"


namespace CCDataStructure {
void createDevicePointCloud(CMesh* vPointCloud,
                            thrust::device_vector<Point>& vDPoints) {
  thrust::host_vector<Point> hPoints;

  for (const auto& item : vPointCloud->vertices) {
    Point p;
    p.position.x = item.Position[0];
    p.position.y = item.Position[1];
    p.position.z = item.Position[2];
    p.normal.x = item.Normal[0];
    p.normal.y = item.Normal[1];
    p.normal.z = item.Normal[2];
    hPoints.push_back(p);
  }

  vDPoints = hPoints;
}

thrust::device_vector<glm::vec4> createDeviceVectorGLM4(int vNum) {
  return thrust::device_vector<glm::vec4>(vNum);
}

thrust::device_vector<float3> createDeviceVectorGLM3(int vNum) {
  return thrust::device_vector<float3>(vNum);
}

thrust::device_vector<float> createDeviceVectorFloat(int vNum) {
  return thrust::device_vector<float>(vNum);
}

DBVHAccel* createDBVHAccel(const ACCEL::BVHAccel* bvhTree) {
  DBVHAccel* hDBVHAccel = new DBVHAccel();

  hDBVHAccel->numTriangles = bvhTree->orderedTriangles.size();
  hDBVHAccel->numNodes = bvhTree->totalLinearNodes;
  CUDACHECKERROR(
      hipMalloc((void**)&(hDBVHAccel->dBVHNodesPointer),
                 sizeof(ACCEL::LinearBVHNode) * bvhTree->totalLinearNodes));
  CUDACHECKERROR(hipMalloc((void**)&hDBVHAccel->dTrianglesPointer,
                            sizeof(Tri) * bvhTree->orderedTriangles.size()));

  CUDACHECKERROR(
      hipMemcpy(hDBVHAccel->dBVHNodesPointer, &bvhTree->nodes[0],
                 sizeof(ACCEL::LinearBVHNode) * bvhTree->totalLinearNodes,
                 hipMemcpyHostToDevice));
  CUDACHECKERROR(hipMemcpy(
      hDBVHAccel->dTrianglesPointer, &bvhTree->orderedTriangles[0],
      sizeof(Tri) * bvhTree->orderedTriangles.size(), hipMemcpyHostToDevice));

  DBVHAccel* dDBVHAccel;
  CUDACHECKERROR(hipMalloc((void**)&dDBVHAccel, sizeof(DBVHAccel)));
  CUDACHECKERROR(hipMemcpy(dDBVHAccel, hDBVHAccel, sizeof(DBVHAccel),
                            hipMemcpyHostToDevice));

  return dDBVHAccel;
}

__device__ bool d_intersect(const DBVHAccel* vBVHPointer, Ray& ray,
                            SurfaceInteraction* isect) {
  const ACCEL::LinearBVHNode* nodes = vBVHPointer->dBVHNodesPointer;
  const Tri* tris = vBVHPointer->dTrianglesPointer;
  const int numNodes = vBVHPointer->numNodes;
  const int numTriangles = vBVHPointer->numTriangles;
  bool hit = false;
  glm::vec3 invDir(glm::min(1 / ray.d.x, 99999.0f),
                   glm::min(1 / ray.d.y, 99999.0f),
                   glm::min(1 / ray.d.z, 99999.0f));
  int dirIsNeg[3] = {invDir[0] < 0, invDir[1] < 0, invDir[2] < 0};
  // Follow ray through BVH nodes to find primitive intersections
  int toVisitOffset = 0, currentNodeIndex = 0;
  int nodesToVisit[6400];

  while (true) {
    if (currentNodeIndex >= numNodes) {
      printf("Wrong Node %d\n", currentNodeIndex);
      return false;
    }
    const ACCEL::LinearBVHNode* node = &nodes[currentNodeIndex];
    // Check ray against BVH node
    float a, b;
    if (node->bounds.Intersect(ray, &a, &b)) {
      if (node->nObject > 0) {
        // Intersect ray with primitives in leaf BVH node
        for (int i = 0; i < node->nObject; ++i) {
          if (node->objectOffset + i >= numTriangles) {
            printf("Wrong tri %d\n", node->objectOffset + i);
            return false;
          }
          if (tris[node->objectOffset + i].Intersect(ray, isect)) hit = true;

        }
        if (toVisitOffset == 0) break;
        currentNodeIndex = nodesToVisit[--toVisitOffset];
      } else {
        if (dirIsNeg[node->axis]) {
          nodesToVisit[toVisitOffset++] = currentNodeIndex + 1;
          currentNodeIndex = node->secondChildOffset;
        } else {
          nodesToVisit[toVisitOffset++] = node->secondChildOffset;
          currentNodeIndex = currentNodeIndex + 1;
        }
      }
    } else {
      if (toVisitOffset == 0) break;
      currentNodeIndex = nodesToVisit[--toVisitOffset];
    }
  }
  return hit;
}

__device__ bool d_visible(const DBVHAccel* vBVHPointer,
                          const glm::vec3 vCameraPos,
                          const glm::vec3 vVertexPosition,
                          const float margin) {
  Ray ray(vCameraPos, vVertexPosition - vCameraPos);
  float current_t = glm::length(vVertexPosition - vCameraPos);
  SurfaceInteraction isect;
  if (!d_intersect(vBVHPointer, ray, &isect)) return false;
  if (current_t <= isect.t) {
    return true;
  }
  return false;
}

}  // namespace CCDataStructure